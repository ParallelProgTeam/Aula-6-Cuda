#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Template project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>
#include <hip/hip_runtime.h>
#include <timer.h>               // timing functions

// CUDA helper functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

StopWatchInterface *timer = NULL;

// includes, kernels
//#include <test1_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel template for flops test
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel( float* g_idata, float* g_odata) 
{
    float result=1;
    // read two values
    float val1 = g_idata[0];
    float val2 = g_idata[1];
 
    // place loop/unrolled loop here to do a bunch of multiply add ops
    // make sure you use results, so compiler does not optomize out
    result = val2 + (result * val1);

     g_odata[0] = result;
}

void cleanup(void)
{
    sdkDeleteTimer(&timer);
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return 0;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{

    hipDeviceProp_t deviceProps;
    float elapsedTimeInMs = 0.0f;
    hipEvent_t start, stop;
    sdkCreateTimer(&timer);
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    //unsigned int timer = 0;
    int devID = findCudaDevice(argc, (const char **)argv);

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors\n",
           deviceProps.name, deviceProps.multiProcessorCount);


//    CUT_SAFE_CALL( cutCreateTimer( &timer));
//    CUT_SAFE_CALL( cutStartTimer( timer));

    // adjust number of threads here
    unsigned int num_threads = 2;
    unsigned int mem_size = sizeof( float) * num_threads;

    sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    // allocate host memory
    float* h_idata = (float*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float* d_idata;
    hipMalloc( (void**) &d_idata, mem_size);
    // copy host memory to device
    checkCudaErrors(hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice)) ;

    // allocate device memory for result
    float* d_odata;
    checkCudaErrors(hipMalloc( (void**) &d_odata, mem_size));

    // setup execution parameters
    // adjust thread block sizes here
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>( d_idata, d_odata);

    // check if kernel execution generated and error

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy( h_odata, d_odata, sizeof( float) * num_threads,
                                hipMemcpyDeviceToHost)) ;

    checkCudaErrors(hipEventRecord(stop, 0));
    //Since device to device memory copies are non-blocking,
    //hipDeviceSynchronize() is required in order to get
    //proper timing.
    checkCudaErrors(hipDeviceSynchronize());

    sdkStopTimer(&timer);
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    printf( "Processing time: %f (ms)\n", elapsedTimeInMs);

    sdkDeleteTimer(&timer);
    checkCudaErrors(hipEventDestroy(stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));
    // cleanup memory
    free( h_idata);
    free( h_odata);
    hipFree(d_idata);
    hipFree(d_odata);
}
